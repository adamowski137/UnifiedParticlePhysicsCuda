#include "hip/hip_runtime.h"
#include "DistanceConstrain.cuh"
#include <cmath>
#include "../Constrain.cuh"

DistanceConstrain::DistanceConstrain(float d, int* indexes) : Constrain{ 2, 1.0f, -10000.0, 10000.0, indexes}, d{d}
{
}

float DistanceConstrain::operator()(float* x, float* y, float* z,
	float* vx, float* vy, float* vz)
{
	int p1 = dev_indexes[0];
	int p2 = dev_indexes[1];
	float distX = (x[p1] - x[p2]) * (x[p1] * x[p2]);
	float distY = (y[p1] - y[p2]) * (y[p1] * y[p2]);
	float distZ = (z[p1] - z[p2]) * (z[p1] * z[p2]);

	return sqrtf(distX + distY + distZ) - d;
}

float DistanceConstrain::timeDerivative(float* x, float* y, float* z,
	float* vx, float* vy, float* vz)
{
	int p1 = dev_indexes[0];
	int p2 = dev_indexes[1];
	float distX = (vx[p1] - vx[p2]) * (vx[p1] * vx[p2]);
	float distY = (vy[p1] - vy[p2]) * (vy[p1] * vy[p2]);
	float distZ = (vz[p1] - vz[p2]) * (vz[p1] * vz[p2]);

	return sqrtf(distX + distY + distZ);
}

float DistanceConstrain::positionDerivative(float* x, float* y, float* z,
	float* vx, float* vy, float* vz, int index)
{
	return 1.0f;
}

float DistanceConstrain::timePositionDerivative(float* x, float* y, float* z,
	float* vx, float* vy, float* vz, int index)
{
	return 0.0f;
}




