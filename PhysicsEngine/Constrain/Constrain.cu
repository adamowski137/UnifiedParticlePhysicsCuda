#include "Constrain.cuh"
#include <hip/hip_runtime.h>
#include "../GpuErrorHandling.hpp"

Constrain::Constrain(int n, float k, float cMin, float cMax, int* indexes) : n{n}, k{k}, cMin{cMin}, cMax{cMax}
{
	gpuErrchk(hipMalloc((void**)&dev_indexes, n * sizeof(int)));
	gpuErrchk(hipMemcpy(dev_indexes, indexes, n * sizeof(int), hipMemcpyHostToDevice));
}

Constrain::~Constrain()
{
	gpuErrchk(hipFree(dev_indexes));
}
