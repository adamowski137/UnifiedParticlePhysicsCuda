#include "Constrain.cuh"
#include <hip/hip_runtime.h>
#include "../GpuErrorHandling.hpp"
#include <iostream>
#include <climits>

Constrain::Constrain(int n, float k, ConstraintLimitType type) : n{n}, k{k}
{
	switch (type)
	{
	case ConstraintLimitType::EQ:
		cMax = FLT_MAX;
		cMin = FLT_MIN;
		break;
	case ConstraintLimitType::GEQ:
		cMax = FLT_MAX;
		cMin = 0;
		break;
	case ConstraintLimitType::LEQ:
		cMax = 0;
		cMin = FLT_MIN;
		break;
	default:
		break;
	}

	//gpuErrchk(hipMalloc((void**)&dev_indexes, n * sizeof(int)));
	//gpuErrchk(hipMemcpy(dev_indexes, indexes, n * sizeof(int), hipMemcpyHostToDevice));
	//int tmp[2];
	//gpuErrchk(hipMemcpy(tmp, dev_indexes, n * sizeof(int), hipMemcpyDeviceToHost));
}

Constrain::~Constrain()
{
	// MEMORY LEAK!!! 
	//gpuErrchk(hipFree(dev_indexes));
}
