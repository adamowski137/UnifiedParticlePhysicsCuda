#include "Constrain.cuh"
#include <hip/hip_runtime.h>
#include "../GpuErrorHandling.hpp"
#include <iostream>

Constrain::Constrain(int n, float k, float cMin, float cMax, int* indexes) : n{n}, k{k}, cMin{cMin}, cMax{cMax}
{
	gpuErrchk(hipMalloc((void**)&dev_indexes, n * sizeof(int)));
	gpuErrchk(hipMemcpy(dev_indexes, indexes, n * sizeof(int), hipMemcpyHostToDevice));
	//int tmp[2];
	//gpuErrchk(hipMemcpy(tmp, dev_indexes, n * sizeof(int), hipMemcpyDeviceToHost));
}

Constrain::~Constrain()
{
	// MEMORY LEAK!!! 
	//gpuErrchk(hipFree(dev_indexes));
}
