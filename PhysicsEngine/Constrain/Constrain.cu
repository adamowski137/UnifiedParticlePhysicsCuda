#include "Constrain.cuh"
#include <hip/hip_runtime.h>
#include "../GpuErrorHandling.hpp"
#include <iostream>
#include <climits>

__host__ __device__ Constrain::Constrain(int n, float k, ConstraintLimitType type) : n{n}, k{k}
{
	switch (type)
	{
	case ConstraintLimitType::EQ:
		cMax = FLT_MAX;
		cMin = FLT_MIN;
		break;
	case ConstraintLimitType::GEQ:
		cMax = FLT_MAX;
		cMin = 0;
		break;
	case ConstraintLimitType::LEQ:
		cMax = 0;
		cMin = FLT_MIN;
		break;
	default:
		break;
	}

	//gpuErrchk(hipMalloc((void**)&dev_indexes, n * sizeof(int)));
	//gpuErrchk(hipMemcpy(dev_indexes, indexes, n * sizeof(int), hipMemcpyHostToDevice));
	//int tmp[2];
	//gpuErrchk(hipMemcpy(tmp, dev_indexes, n * sizeof(int), hipMemcpyDeviceToHost));
}
