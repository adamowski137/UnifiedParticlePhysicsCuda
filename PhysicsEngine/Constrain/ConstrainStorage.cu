#include "hip/hip_runtime.h"
#include "ConstrainStorage.cuh"
#include <>
#include <hip/hip_runtime.h>

__global__ void addCollisionsKern(List* collisions, int* counts, DistanceConstrain* constraints, ConstraintLimitType type, float d, int nParticles)
{
	const int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= nParticles - 1) return;
	Node* p = collisions[index].head;
	int constrainIndex = counts[index] - 1;

	while (p != NULL)
	{
		constraints[constrainIndex] = DistanceConstrain().init(d, index, p->value, type);
		p = p->next;
		constrainIndex--;
	}
}

__device__ __constant__ DistanceConstrain CUDAConstants::staticDistanceConstraints[MAX_CONSTRAINS];
__device__ __constant__ SurfaceConstraint CUDAConstants::staticSurfaceConstraints[MAX_CONSTRAINS];

ConstrainStorage ConstrainStorage::Instance;



void ConstrainStorage::initInstance()
{
	gpuErrchk(hipMalloc((void**)&dynamicDistanceConstraints, DEFAULT_CONSTRAINS * sizeof(DistanceConstrain)));
	gpuErrchk(hipMalloc((void**)&dynamicSurfaceConstraints, DEFAULT_CONSTRAINS * sizeof(DistanceConstrain)));
	
	for (int i = 0; i < CONSTRAINTYPESNUMBER; i++)
	{
		nStaticConstraints[i] = 0;
		nDynamicConstraints[i] = 0;
		maxDynamicConstraints[i] = DEFAULT_CONSTRAINS;
	}
}

ConstrainStorage::~ConstrainStorage()
{
	gpuErrchk(hipFree(dynamicDistanceConstraints));
	gpuErrchk(hipFree(dynamicSurfaceConstraints));
}


void ConstrainStorage::addCollisions(List* collisions, int* counts, ConstraintLimitType type, float d, int nParticles)
{
	nDynamicConstraints[(int)ConstrainType::DISTANCE] = counts[nParticles - 1];
	if (maxDynamicConstraints[(int)ConstrainType::DISTANCE] < counts[nParticles - 1])
	{
		maxDynamicConstraints[(int)ConstrainType::DISTANCE] = counts[nParticles - 1];
		gpuErrchk(hipFree(dynamicDistanceConstraints));
		gpuErrchk(hipMalloc((void**)&dynamicDistanceConstraints, counts[nParticles - 1] * sizeof(DistanceConstrain)));
	}

	unsigned int threads = 32;
	int particle_bound_blocks = (nParticles + threads - 1) / threads;

	addCollisionsKern< <<particle_bound_blocks, threads> >>(collisions, counts, dynamicDistanceConstraints, type, d, nParticles);
	gpuErrchk(hipGetLastError());
	gpuErrchk(hipDeviceSynchronize());
}
