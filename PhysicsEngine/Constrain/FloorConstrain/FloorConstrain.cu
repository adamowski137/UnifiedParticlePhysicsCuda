#include "hip/hip_runtime.h"
#include "FloorConstrain.cuh"
#include <cmath>
#include "../Constrain.cuh"

FloorConstrain::FloorConstrain(float d, int* indexes) : Constrain{ 2, 1.0f, -10000.0, 10000.0, indexes }, d{ d }
{
}

float FloorConstrain::operator()(float* x, float* y, float* z,
	float* vx, float* vy, float* vz)
{
	int p1 = dev_indexes[0];
	int p2 = dev_indexes[1];
	float distX = (x[p1] - x[p2]) * (x[p1] - x[p2]);
	float distY = (y[p1] - y[p2]) * (y[p1] - y[p2]);
	float distZ = (z[p1] - z[p2]) * (z[p1] - z[p2]);

	return sqrtf(distX + distY + distZ) - d;
}

float FloorConstrain::timeDerivative(float* x, float* y, float* z,
	float* vx, float* vy, float* vz)
{
	int p1 = dev_indexes[0];
	int p2 = dev_indexes[1];

	float distX = (x[p1] - x[p2]) * (x[p1] - x[p2]);
	float distY = (y[p1] - y[p2]) * (y[p1] - y[p2]);
	float distZ = (z[p1] - z[p2]) * (z[p1] - z[p2]);

	float nx = (x[p1] - x[p2]);
	float ny = (y[p1] - y[p2]);
	float nz = (z[p1] - z[p2]);

	float len = sqrt(nx * nx + ny * ny + nz * nz);

	nx /= len;
	ny /= len;
	nz /= len;

	float diffvX = (vx[p1] - vx[p2]);
	float diffvY = (vy[p1] - vy[p2]);
	float diffvZ = (vz[p1] - vz[p2]);

	//return diffvX + diffvY + diffvZ;

	//return sqrt(diffvX * diffvX + diffvY * diffvY + diffvZ * diffvZ);
	float coeff = 1 / sqrtf(distX + distY + distZ);
	//return coeff > 1.f ? 1.f : coeff * (diffvX + diffvY + diffvZ);
	return coeff * (nx * vx[p1] + ny * vy[p1] + nz * vz[p1] - nx * vx[p2] - ny * vy[p2] - nz * vz[p2]);
}

void FloorConstrain::positionDerivative(float* x, float* y, float* z,
	float* vx, float* vy, float* vz, int index, float* output)
{
	int p1 = dev_indexes[0];
	int p2 = dev_indexes[1];
	//float len = sqrtf((x[p1] - x[p2]) * (x[p1] - x[p2]) + (y[p1] - y[p2]) * (y[p1] - y[p2]) + (z[p1] - z[p2]) * (z[p1] - z[p2]));
	if (index == 0)
	{
		output[0] = x[p1] - x[p2];
		output[1] = y[p1] - y[p2];
		output[2] = z[p1] - z[p2];
	}
	else
	{
		output[0] = x[p2] - x[p1];
		output[1] = y[p2] - y[p1];
		output[2] = z[p2] - z[p1];
	}
}

void FloorConstrain::timePositionDerivative(float* x, float* y, float* z,
	float* vx, float* vy, float* vz, int index, float* output)
{
	output[0] = 0.0f;
	output[1] = 0.0f;
	output[2] = 0.0f;
}
