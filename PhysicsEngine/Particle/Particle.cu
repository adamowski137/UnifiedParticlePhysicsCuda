#include "hip/hip_runtime.h"
#include <glad/glad.h>
#include "Particle.cuh"
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <cuda_gl_interop.h>
#include <>
#include <memory>
#include <thrust/fill.h>
#include <thrust/device_ptr.h>
#include "../GpuErrorHandling.hpp"
#include "../Constrain/DistanceConstrain/DistanceConstrain.cuh"
#include "../Constants.hpp"

#define EPS 0.000001

__global__ void initializeRandomKern(int amount, hiprandState* state)
{
	const int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= amount) return;
	hiprand_init(1234, index, 0, &state[index]);
}

__global__ void fillRandomKern(int amount, float* dst, hiprandState* state, float min, float max)
{
	const int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= amount) return;
	dst[index] = (max - min) * hiprand_uniform(&state[index]) + min;
}

__global__ void copyToVBOKernel(int amount, float* x, float* y, float* z, float* dst)
{
	const int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= amount) return;
	dst[3 * index + 0] = x[index];
	dst[3 * index + 1] = y[index];
	dst[3 * index + 2] = z[index];
}

__global__ void setDiagonalMatrix(int amount, float* src, float* dst)
{
	const int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= amount) return;
	dst[amount * index + index] = src[index];
}

__global__ void predictPositionsKern(int amount, 
	float* x, float* y, float* z,
	float* new_x, float* new_y, float* new_z, 
	float* vx, float* vy, float* vz,
	float dvx, float dvy, float dvz,
	float dt
)
{
	const int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= amount) return;

	// update velocities
	vx[index] += dvx;
	vy[index] += dvy;
	vz[index] += dvz;

	// predict new position - not the actual new positions
	new_x[index] = x[index] + dt * vx[index];
	new_y[index] = y[index] + dt * vy[index];
	new_z[index] = z[index] + dt * vz[index];

	// apply mass scaling??
}

__global__ void applyChangesKern(int amount,
	float* x, float* y, float* z,
	float* new_x, float* new_y, float* new_z,
	float* vx, float* vy, float* vz, float* fc, float invdt)
{
	const int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= amount) return;

	float changeX = (new_x[index] - x[index]);
	float changeY = (new_y[index] - y[index]);
	float changeZ = (new_z[index] - z[index]);

	// update velocity
	vx[index] = invdt * (changeX) + fc[3 * index];
	vy[index] = invdt * (changeY) + fc[3 * index + 1];
	vz[index] = invdt * (changeZ) + fc[3 * index + 2];

	// advect diffuse particles ??
	
	// apply internal forces

	// update position or apply sleeping

	float changeSQ = changeX * changeX + changeY * changeY + changeZ * changeZ;
	if (changeSQ > EPS)
	{
		x[index] = new_x[index];
		y[index] = new_y[index];
		z[index] = new_z[index];
	}

}

ParticleType::ParticleType(int amount) : nParticles{amount}
{
	blocks = ceilf((float)nParticles / THREADS);
	constrainSolver = std::unique_ptr<ConstrainSolver>{ new ConstrainSolver{amount}};
	collisionGrid = std::unique_ptr<CollisionGrid>{ new CollisionGrid{amount}};
	setupDeviceData();
}

ParticleType::~ParticleType()
{
	gpuErrchk(hipFree(dev_x));
	gpuErrchk(hipFree(dev_y));
	gpuErrchk(hipFree(dev_z));
	gpuErrchk(hipFree(dev_new_x));
	gpuErrchk(hipFree(dev_new_y));
	gpuErrchk(hipFree(dev_new_z));
	gpuErrchk(hipFree(dev_vx));
	gpuErrchk(hipFree(dev_vy));
	gpuErrchk(hipFree(dev_vz));
	gpuErrchk(hipFree(dev_invmass));
}

void ParticleType::setupDeviceData()
{
	gpuErrchk(hipMalloc((void**)&dev_curand, nParticles * sizeof(hiprandState)));
	gpuErrchk(hipMalloc((void**)&dev_x, nParticles * sizeof(float)));
	gpuErrchk(hipMalloc((void**)&dev_y, nParticles * sizeof(float)));
	gpuErrchk(hipMalloc((void**)&dev_z, nParticles * sizeof(float)));

	gpuErrchk(hipMalloc((void**)&dev_new_x, nParticles * sizeof(float)));
	gpuErrchk(hipMalloc((void**)&dev_new_y, nParticles * sizeof(float)));
	gpuErrchk(hipMalloc((void**)&dev_new_z, nParticles * sizeof(float)));
	
	gpuErrchk(hipMalloc((void**)&dev_vx, nParticles * sizeof(float)));
	gpuErrchk(hipMalloc((void**)&dev_vy, nParticles * sizeof(float)));
	gpuErrchk(hipMalloc((void**)&dev_vz, nParticles * sizeof(float)));
	
	gpuErrchk(hipMalloc((void**)&dev_collisions, nParticles * sizeof(List)));
	gpuErrchk(hipMalloc((void**)&dev_sums, nParticles * sizeof(int)));


	
	gpuErrchk(hipMalloc((void**)&dev_fc, 3 * nParticles * sizeof(float)));

	gpuErrchk(hipMalloc((void**)&dev_invmass, nParticles * sizeof(float)));
	thrust::device_ptr<float> massptr{ dev_invmass };
	thrust::fill(massptr, massptr + nParticles, 1);

	int amountOfConstrains = nParticles;

	initializeRandomKern << < blocks, THREADS >> > (nParticles, dev_curand);
	gpuErrchk(hipGetLastError());
	gpuErrchk(hipDeviceSynchronize());

	fillRandomKern << <blocks, THREADS >> > (nParticles, dev_x, dev_curand, MINDIMENSION, MAXDIMENSION);
	gpuErrchk(hipGetLastError());
	gpuErrchk(hipDeviceSynchronize());

	fillRandomKern << <blocks, THREADS >> > (nParticles, dev_y, dev_curand, MINDIMENSION, MAXDIMENSION);
	gpuErrchk(hipGetLastError());
	gpuErrchk(hipDeviceSynchronize());
	
	fillRandomKern << <blocks, THREADS >> > (nParticles, dev_z, dev_curand, 0.f, 0.f);
	gpuErrchk(hipGetLastError());
	gpuErrchk(hipDeviceSynchronize());

	fillRandomKern << <blocks, THREADS >> > (nParticles, dev_vx, dev_curand, -1.f, 1.f);
	gpuErrchk(hipGetLastError());
	gpuErrchk(hipDeviceSynchronize());
	
	fillRandomKern << <blocks, THREADS >> > (nParticles, dev_vy, dev_curand, -1.f, 1.f);
	gpuErrchk(hipGetLastError());
	gpuErrchk(hipDeviceSynchronize());
	
	//fillRandomKern << <blocks, THREADS >> > (nParticles, dev_vz, dev_curand, -5.f, 5.f);
	//gpuErrchk(hipGetLastError());
	//gpuErrchk(hipDeviceSynchronize());
}

void ParticleType::renderData(unsigned int vbo)
{
	float* dst;
	cudaGLMapBufferObject((void**)&dst, vbo);

	copyToVBOKernel <<<blocks, THREADS>>>(nParticles, dev_x, dev_y, dev_z, dst);

	gpuErrchk(hipGetLastError());
	gpuErrchk(hipDeviceSynchronize());
	
	cudaGLUnmapBufferObject(vbo);
}

void ParticleType::calculateNewPositions(float dt)
{
	hipMemset(dev_fc, 0, 3 * nParticles * sizeof(float));
	// predict new positions and update velocities
	fextx = 0.0f;
	//fexty = -9.81f;
	fexty = 0.0f;
	fextz = 0.0f;

	float dvx = fextx * dt;
	float dvy = fexty * dt;
	float dvz = fextz * dt;

	predictPositionsKern << <blocks, THREADS >> > (
		nParticles,
		dev_x, dev_y, dev_z,
		dev_new_x, dev_new_y, dev_new_z,
		dev_vx, dev_vy, dev_vz,
		dvx, dvy, dvz, dt
		);

	gpuErrchk(hipGetLastError());
	gpuErrchk(hipDeviceSynchronize());

	// find neighboring particles and solid contacts ??

	collisionGrid->findCollisions(dev_x, dev_y, dev_z, nParticles, dev_sums, dev_collisions);

	// todo implement grid (predicted positions)

	// stabilization iterations

	gpuErrchk(hipGetLastError());
	gpuErrchk(hipDeviceSynchronize());

	// todo solve contact constrains
	// update predicted position and current positions

	// solve iterations
	constrainSolver->addDynamicConstraints(dev_collisions, dev_sums, PARTICLERADIUS, ConstraintLimitType::GEQ);
	constrainSolver->calculateForces(dev_new_x, dev_new_y, dev_new_z, dev_vx, dev_vy, dev_vz, dev_invmass, dev_fc, dt);

	// todo solve every constraint group 
	// update predicted position
	applyChangesKern << <blocks, THREADS >> > (
		nParticles,
		dev_x, dev_y, dev_z,
		dev_new_x, dev_new_y, dev_new_z,
		dev_vx, dev_vy, dev_vz, dev_fc,
		1/dt
		);
	gpuErrchk(hipGetLastError());
	gpuErrchk(hipDeviceSynchronize());
}

void ParticleType::setConstraints(std::vector<std::pair<int, int>> pairs, float d)
{
	this->constrainSolver->setStaticConstraints(pairs, d);
}

void ParticleType::mapCudaVBO(unsigned int vbo)
{
	cudaGLRegisterBufferObject(vbo);
}