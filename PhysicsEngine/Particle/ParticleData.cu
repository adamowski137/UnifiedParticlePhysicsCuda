#include "hip/hip_runtime.h"
#include "ParticleData.cuh"
#include <>

__global__ void initializeRandomKern(int amount, hiprandState* state)
{
	const int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= amount) return;
	hiprand_init(1234, index, 0, &state[index]);
}

__global__ void fillRandomKern(int amount, float* dst, hiprandState* state, float min, float max)
{
	const int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= amount) return;
	dst[index] = (max - min) * hiprand_uniform(&state[index]) + min;
}
