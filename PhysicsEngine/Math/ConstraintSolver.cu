#include "hip/hip_runtime.h"
#include "ConstraintSolver.cuh"
#include "../GpuErrorHandling.hpp"
#include <hip/hip_runtime.h>
#include <>
#include <thrust/transform.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include "../Constants.hpp"

#define SHMEM_SIZE 1024


template<typename T>
__global__ void fillJacobiansKern(
	int nConstraints, int nParticles,
	float* x, float* y, float* z,
	float* jacobian,
	T* constrains, ConstraintType type)
{
	const int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= nConstraints) return;
	if (type == ConstraintType::DISTANCE)
	{
		(constrains[index]).positionDerivative(x, y, z, 0, &jacobian[index * 3 * nParticles + 3 * (constrains[index]).p[0]]);
		(constrains[index]).positionDerivative(x, y, z, 1, &jacobian[index * 3 * nParticles + 3 * (constrains[index]).p[1]]);
	}
	if (type == ConstraintType::SURFACE)
	{
		(constrains[index]).positionDerivative(x, y, z, 0, &jacobian[index * 3 * nParticles + 3 * (constrains[index]).p[0]]);
	}
}


template <typename T>
__global__ void fillResultVectorKern(int particles, int constrainsNumber, float* b,
	float* x, float* y, float* z,
	float* jacobian,
	float* dev_c_min, float* dev_c_max,
	T* constrains, float dt)
{
	const int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= constrainsNumber) return;
	b[index] = -(constrains[index])(x, y, z, dt);
	dev_c_max[index] = constrains[index].cMax;
	dev_c_min[index] = constrains[index].cMin;
}


__global__ void matrixMulKern(const float* a, const float* b, float* c, int N, int K) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ float s_a[SHMEM_SIZE];
	__shared__ float s_b[SHMEM_SIZE];

	float tmp = 0;

	for (int i = 0; i < K; i += blockDim.x)
	{

		s_a[threadIdx.y * blockDim.x + threadIdx.x] = 0;
		s_b[threadIdx.y * blockDim.x + threadIdx.x] = 0;
		__syncthreads();


		if (row < N && i + threadIdx.x < K)
			s_a[threadIdx.y * blockDim.x + threadIdx.x] = a[row * K + i + threadIdx.x];
		if (col < N && i + threadIdx.y < K)
			s_b[threadIdx.y * blockDim.x + threadIdx.x] = b[i * N + threadIdx.y * N + col];
		__syncthreads();

		if (row < N && col < N)
		{
			for (int j = 0; j < blockDim.x; j++) {
				tmp += s_a[threadIdx.y * blockDim.x + j] * s_b[j * blockDim.x + threadIdx.x];
			}
		}
		__syncthreads();
	}

	if (row < N && col < N)
		c[row * N + col] = tmp;
}

__global__ void massVectorMultpilyKern(int columns, int rows, float* invMass, float* J)
{
	const int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= columns * rows) return;
	int column = index % columns;
	J[index] *= invMass[column / 3];
}

__global__ void transposeKern(int columns, int rows, float* A, float* AT)
{
	const int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= columns * rows) return;
	int column = index % columns;
	int row = index / columns;

	AT[column * rows + row] = A[row * columns + column];
}


__global__ void applyForce(float* new_lambda, float* jacobi_transposed, float* dx, float* dy, float* dz, float dt, int nParticles, int nConstraints)
{
	const int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index < nParticles)
	{
		for (int i = 0; i < nConstraints; i++)
		{
			float sumX = 0, sumY = 0, sumZ = 0;
			int sumC = 0;
			for (int i = 0; i < nConstraints; i++)
			{
				sumC++;
				sumX += new_lambda[i] * jacobi_transposed[(3 * index + 0) * nConstraints + i];
				sumY += new_lambda[i] * jacobi_transposed[(3 * index + 1) * nConstraints + i];
				sumZ += new_lambda[i] * jacobi_transposed[(3 * index + 2) * nConstraints + i];
			}
			if (sumC == 0 || index == 0 || index == 2) return;
			//dx[index] += 1.5f * sumX * dt / sumC;
			//dy[index] += 1.5f * sumY * dt / sumC;
			//dz[index] += 1.5f * sumZ * dt / sumC;

			dx[index] += sumX * dt;
			dy[index] += sumY * dt;
			dz[index] += sumZ * dt;

		}
	}
}

template<typename T>
void fillJacobiansWrapper(int nConstraints, int nParticles,
	float* x, float* y, float* z,
	float* dx, float* dy, float* dz,
	float* jacobian,
	float* jacobian_transposed, float* A,
	float* b, float dt,
	float* invmass, float* lambda, float* new_lambda, float* c_min, float* c_max,
	T* constraints, ConstraintType type, int iterations)
{
	unsigned int threads = 32;

	// kernels bound by number of constraints
	int constraint_bound_blocks = (nConstraints + threads - 1) / threads;

	// kernels bound by the size of Jacobian
	int jacobian_bound_blocks = ((3 * nParticles * nConstraints) + threads - 1) / threads;

	int particlex3_bound_blocks = ((3 * nParticles) + threads - 1) / threads;

	int particle_bound_blocks = (nParticles + threads - 1) / threads;

	fillJacobiansKern << <constraint_bound_blocks, threads >> > (nConstraints, nParticles,
		x, y, z,
		jacobian,
		constraints, type);

	gpuErrchk(hipGetLastError());
	gpuErrchk(hipDeviceSynchronize());

	transposeKern << <jacobian_bound_blocks, threads >> > (
		3 * nParticles,
		nConstraints,
		jacobian,
		jacobian_transposed);


	fillResultVectorKern << <constraint_bound_blocks, threads >> > (nParticles, nConstraints, b,
		x, y, z,
		jacobian,
		c_min, c_max,
		constraints, dt);

	gpuErrchk(hipGetLastError());
	gpuErrchk(hipDeviceSynchronize());
	massVectorMultpilyKern << <jacobian_bound_blocks, threads >> > (
		3 * nParticles,
		nConstraints,
		invmass,
		jacobian);

	gpuErrchk(hipGetLastError());
	gpuErrchk(hipDeviceSynchronize());

	unsigned int BLOCKS_X = (nConstraints + threads - 1) / threads;
	unsigned int BLOCKS_Y = (nConstraints + threads - 1) / threads;

	dim3 th{ threads, threads };
	dim3 bl{ BLOCKS_X, BLOCKS_Y };

	matrixMulKern << <bl, th >> > (jacobian, jacobian_transposed, A, nConstraints, 3 * nParticles);

	gpuErrchk(hipGetLastError());
	gpuErrchk(hipDeviceSynchronize());

	jaccobi(nConstraints, A, b, lambda, new_lambda, c_min, c_max, iterations);

	applyForce << <particle_bound_blocks, threads >> > (new_lambda, jacobian_transposed, dx, dy, dz, dt, nParticles, nConstraints);

	gpuErrchk(hipGetLastError());
	gpuErrchk(hipDeviceSynchronize());
}

ConstraintSolver::ConstraintSolver(int particles) : nParticles{ particles }
{
	// set pointers to 0 - this way it will be easy to distinguish whether they have already been allocated or not
	dev_jacobian = 0;
	dev_jacobian_transposed = 0;
	dev_A = 0;
	dev_b = 0;
	dev_lambda = 0;
	dev_new_lambda = 0;
	dev_c_min = 0;
	dev_c_max = 0;

	nConstraintsMaxAllocated = 1;
	this->allocateArrays(50);

	gpuErrchk(hipMalloc((void**)&dev_dx, nParticles * sizeof(float)));
	gpuErrchk(hipMalloc((void**)&dev_dy, nParticles * sizeof(float)));
	gpuErrchk(hipMalloc((void**)&dev_dz, nParticles * sizeof(float)));

	gpuErrchk(hipMemset(dev_dx, 0, nParticles * sizeof(float)));
	gpuErrchk(hipMemset(dev_dy, 0, nParticles * sizeof(float)));
	gpuErrchk(hipMemset(dev_dz, 0, nParticles * sizeof(float)));

	ConstraintStorage::Instance.initInstance();
}

ConstraintSolver::~ConstraintSolver()
{
	gpuErrchk(hipFree(dev_jacobian));
	gpuErrchk(hipFree(dev_jacobian_transposed));
	gpuErrchk(hipFree(dev_A));
	gpuErrchk(hipFree(dev_b));
	gpuErrchk(hipFree(dev_lambda));
	gpuErrchk(hipFree(dev_new_lambda));
	gpuErrchk(hipFree(dev_c_min));
	gpuErrchk(hipFree(dev_c_max));
	gpuErrchk(hipFree(dev_dx));
	gpuErrchk(hipFree(dev_dy));
	gpuErrchk(hipFree(dev_dz));
}

void ConstraintSolver::calculateForces(
	float* new_x, float* new_y, float* new_z,
	float* invmass, float dt, int iterations
)
{
	int num_iterations = 1;
	for (int i = 0; i < num_iterations; i++)
	{

		gpuErrchk(hipMemset(dev_dx, 0, nParticles * sizeof(float)));
		gpuErrchk(hipMemset(dev_dy, 0, nParticles * sizeof(float)));
		gpuErrchk(hipMemset(dev_dz, 0, nParticles * sizeof(float)));

		thrust::device_ptr<float> thrust_x(new_x);
		thrust::device_ptr<float> thrust_y(new_y);
		thrust::device_ptr<float> thrust_z(new_z);

		thrust::device_ptr<float> thrust_dx(dev_dx);
		thrust::device_ptr<float> thrust_dy(dev_dy);
		thrust::device_ptr<float> thrust_dz(dev_dz);

		this->projectConstraints<SurfaceConstraint>(invmass, new_x, new_y, new_z, dt / num_iterations, ConstraintType::SURFACE, true, iterations);
		this->projectConstraints<DistanceConstraint>(invmass, new_x, new_y, new_z, dt / num_iterations, ConstraintType::DISTANCE, true, iterations);
		this->projectConstraints<DistanceConstraint>(invmass, new_x, new_y, new_z, dt / num_iterations, ConstraintType::DISTANCE, false, iterations);


		thrust::transform(thrust_x, thrust_x + nParticles, thrust_dx, thrust_x, thrust::plus<float>());
		thrust::transform(thrust_y, thrust_y + nParticles, thrust_dy, thrust_y, thrust::plus<float>());
		thrust::transform(thrust_z, thrust_z + nParticles, thrust_dz, thrust_z, thrust::plus<float>());

		for (int i = 0; i < nParticles; i++)
			std::cout << "(" << thrust_dx[i] << ", " << thrust_dy[i] << ", " << thrust_dz[i] << "), ";
		std::cout << "\n";
	}

	//ConstraintStorage::Instance.clearConstraints(); 
}

void ConstraintSolver::calculateStabilisationForces(
	float* x, float* y, float* z,
	float* new_x, float* new_y, float* new_z,
	float* invmass, float dt, int iterations)
{
	gpuErrchk(hipMemset(dev_dx, 0, nParticles * sizeof(float)));
	gpuErrchk(hipMemset(dev_dy, 0, nParticles * sizeof(float)));
	gpuErrchk(hipMemset(dev_dz, 0, nParticles * sizeof(float)));

	thrust::device_ptr<float> thrust_x(x);
	thrust::device_ptr<float> thrust_y(y);
	thrust::device_ptr<float> thrust_z(z);

	thrust::device_ptr<float> thrust_new_x(new_x);
	thrust::device_ptr<float> thrust_new_y(new_y);
	thrust::device_ptr<float> thrust_new_z(new_z);

	thrust::device_ptr<float> thrust_dx(dev_dx);
	thrust::device_ptr<float> thrust_dy(dev_dy);
	thrust::device_ptr<float> thrust_dz(dev_dz);

	this->projectConstraints<DistanceConstraint>(invmass, x, y, z, dt, ConstraintType::DISTANCE, true, iterations);
	this->projectConstraints<SurfaceConstraint>(invmass, x, y, z, dt, ConstraintType::SURFACE, true, iterations);

	thrust::transform(thrust_new_x, thrust_new_x + nParticles, thrust_dx, thrust_new_x, thrust::plus<float>());
	thrust::transform(thrust_new_y, thrust_new_y + nParticles, thrust_dy, thrust_new_y, thrust::plus<float>());
	thrust::transform(thrust_new_z, thrust_new_z + nParticles, thrust_dz, thrust_new_z, thrust::plus<float>());

	thrust::transform(thrust_x, thrust_x + nParticles, thrust_dx, thrust_x, thrust::plus<float>());
	thrust::transform(thrust_y, thrust_y + nParticles, thrust_dy, thrust_y, thrust::plus<float>());
	thrust::transform(thrust_z, thrust_z + nParticles, thrust_dz, thrust_z, thrust::plus<float>());

	//ConstraintStorage::Instance.clearConstraints();
}

void ConstraintSolver::setStaticConstraints(std::vector<std::pair<int, int>> pairs, float d)
{
	std::vector<DistanceConstraint> cpu_constraints;
	for (const auto& pair : pairs)
	{
		cpu_constraints.push_back(DistanceConstraint().init(d, pair.first, pair.second, ConstraintLimitType::EQ));
	}

	ConstraintStorage::Instance.setStaticConstraints<DistanceConstraint>(cpu_constraints.data(), cpu_constraints.size(), ConstraintType::DISTANCE);

}

void ConstraintSolver::addDynamicConstraints(List* collisions, int* sums, float d, ConstraintLimitType type)
{
	ConstraintStorage::Instance.addCollisions(collisions, sums, type, d, nParticles);
}

void ConstraintSolver::addSurfaceConstraints(SurfaceConstraint* surfaceConstraints, int nSurfaceConstraints)
{
	ConstraintStorage::Instance.setDynamicConstraints<SurfaceConstraint>(surfaceConstraints, nSurfaceConstraints, ConstraintType::SURFACE);
}

void ConstraintSolver::allocateArrays(int nConstraints)
{
	if (nConstraints > nConstraintsMaxAllocated)
	{
		while (nConstraints > nConstraintsMaxAllocated)
		{
			nConstraintsMaxAllocated *= 2;
		}

		if (dev_jacobian != 0)
			gpuErrchk(hipFree(dev_jacobian));
		gpuErrchk(hipMalloc((void**)&dev_jacobian, 3 * nParticles * nConstraintsMaxAllocated * sizeof(float)));
		gpuErrchk(hipMemset(dev_jacobian, 0, 3 * nParticles * nConstraintsMaxAllocated * sizeof(float)));

		if (dev_jacobian_transposed != 0)
			gpuErrchk(hipFree(dev_jacobian_transposed));
		gpuErrchk(hipMalloc((void**)&dev_jacobian_transposed, 3 * nParticles * nConstraintsMaxAllocated * sizeof(float)));
		gpuErrchk(hipMemset(dev_jacobian_transposed, 0, 3 * nParticles * nConstraintsMaxAllocated * sizeof(float)));

		if (dev_A != 0)
			gpuErrchk(hipFree(dev_A));
		gpuErrchk(hipMalloc((void**)&dev_A, nConstraintsMaxAllocated * nConstraintsMaxAllocated * sizeof(float)));
		gpuErrchk(hipMemset(dev_A, 0, nConstraintsMaxAllocated * nConstraintsMaxAllocated * sizeof(float)));

		if (dev_b != 0)
			gpuErrchk(hipFree(dev_b));
		gpuErrchk(hipMalloc((void**)&dev_b, nConstraintsMaxAllocated * sizeof(float)));
		gpuErrchk(hipMemset(dev_b, 0, nConstraintsMaxAllocated * sizeof(float)));

		if (dev_lambda != 0)
			gpuErrchk(hipFree(dev_lambda));
		gpuErrchk(hipMalloc((void**)&dev_lambda, nConstraintsMaxAllocated * sizeof(float)));
		gpuErrchk(hipMemset(dev_lambda, 0, nConstraintsMaxAllocated * sizeof(float)));

		if (dev_new_lambda != 0)
			gpuErrchk(hipFree(dev_new_lambda));
		gpuErrchk(hipMalloc((void**)&dev_new_lambda, nConstraintsMaxAllocated * sizeof(float)));
		gpuErrchk(hipMemset(dev_new_lambda, 0, nConstraintsMaxAllocated * sizeof(float)));

		if (dev_c_min != 0)
			gpuErrchk(hipFree(dev_c_min));
		gpuErrchk(hipMalloc((void**)&dev_c_min, nConstraintsMaxAllocated * sizeof(float)));
		gpuErrchk(hipMemset(dev_c_min, 0, nConstraintsMaxAllocated * sizeof(float)));

		if (dev_c_max != 0)
			gpuErrchk(hipFree(dev_c_max));
		gpuErrchk(hipMalloc((void**)&dev_c_max, nConstraintsMaxAllocated * sizeof(float)));
		gpuErrchk(hipMemset(dev_c_max, 0, nConstraintsMaxAllocated * sizeof(float)));

	}
	else this->clearArrays(nConstraints);
}

void ConstraintSolver::clearArrays(int nConstraints)
{
	gpuErrchk(hipMemset(dev_jacobian, 0, 3 * nParticles * nConstraints * sizeof(float)));
	gpuErrchk(hipMemset(dev_jacobian_transposed, 0, 3 * nParticles * nConstraints * sizeof(float)));
	gpuErrchk(hipMemset(dev_b, 0, nConstraints * sizeof(float)));
	gpuErrchk(hipMemset(dev_new_lambda, 0, nConstraints * sizeof(float)));

}
