#include "ConstraintSolver.cuh"
#include "../../Constraint/ConstraintStorage.cuh"

ConstraintSolver::ConstraintSolver(int particles) : nParticles{particles}
{
	gpuErrchk(hipMalloc((void**)&dev_dx, nParticles * sizeof(float)));
	gpuErrchk(hipMalloc((void**)&dev_dy, nParticles * sizeof(float)));
	gpuErrchk(hipMalloc((void**)&dev_dz, nParticles * sizeof(float)));

	gpuErrchk(hipMemset(dev_dx, 0, nParticles * sizeof(float)));
	gpuErrchk(hipMemset(dev_dy, 0, nParticles * sizeof(float)));
	gpuErrchk(hipMemset(dev_dz, 0, nParticles * sizeof(float)));

	ConstraintStorage<DistanceConstraint>::Instance.initInstance();
	ConstraintStorage<SurfaceConstraint>::Instance.initInstance();
}

ConstraintSolver::~ConstraintSolver()
{
	gpuErrchk(hipFree(dev_dx));
	gpuErrchk(hipFree(dev_dy));
	gpuErrchk(hipFree(dev_dz));
}

void ConstraintSolver::clearAllConstraints()
{
	ConstraintStorage<DistanceConstraint>::Instance.clearConstraints();
	ConstraintStorage<SurfaceConstraint>::Instance.clearConstraints();
}
