#include "DirectConstraintSolverCPU.cuh"
#include "../../../Constraint/ConstraintStorage.cuh"
#include "../../../GpuErrorHandling.hpp"
#include "thrust/device_ptr.h"

DirectConstraintSolverCPU::DirectConstraintSolverCPU(int nParticles) : ConstraintSolver(nParticles)
{
	x_cpu = new float[nParticles];
	y_cpu = new float[nParticles];
	z_cpu = new float[nParticles];
	invmass_cpu = new float[nParticles];
}

DirectConstraintSolverCPU::~DirectConstraintSolverCPU()
{
	delete[] x_cpu;
	delete[] y_cpu;
	delete[] z_cpu;
	delete[] invmass_cpu;
}

void DirectConstraintSolverCPU::calculateForces(float dt, int iterations)
{
	auto args = builder.build();
	gpuErrchk(hipMemcpy(x_cpu, args.new_x, sizeof(float) * nParticles, hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(y_cpu, args.new_y, sizeof(float) * nParticles, hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(z_cpu, args.new_z, sizeof(float) * nParticles, hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(invmass_cpu, args.invmass, sizeof(float) * nParticles, hipMemcpyDeviceToHost));
	memset(lambda, 0, sizeof(float) * 2000);

	
	for (int i = 0; i < iterations; i++)
	{
		this->projectConstraints<DistanceConstraint>(args.new_x, args.new_y, args.new_z, args.invmass, dt / iterations, iterations);
		this->projectConstraints<SurfaceConstraint>(args.new_x, args.new_y, args.new_z, args.invmass, dt / iterations, iterations);
	}

	gpuErrchk(hipMemcpy(args.new_x, x_cpu, sizeof(float) * nParticles, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(args.new_y, y_cpu, sizeof(float) * nParticles, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(args.new_z, z_cpu, sizeof(float) * nParticles, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(args.invmass, invmass_cpu, sizeof(float) * nParticles, hipMemcpyHostToDevice));

	clearAllConstraints();
}

void DirectConstraintSolverCPU::calculateStabilisationForces(float dt, int iterations)
{
	throw - 1;
}

template<typename T>
void DirectConstraintSolverCPU::projectConstraints(float* x, float* y, float* z, float* invmass, float dt, int iterations)
{

	auto constraintData = ConstraintStorage<T>::Instance.getConstraints();
	T* cpu_constraints = new T[constraintData.second];
	//std::cout << n << "\n";

	gpuErrchk(hipMemcpy(cpu_constraints, constraintData.first, sizeof(T) * constraintData.second, hipMemcpyDeviceToHost));

	for (int i = 0; i < constraintData.second; i++)
	{
		cpu_constraints[i].directSolve_cpu(x_cpu, y_cpu, z_cpu, invmass_cpu);
	}

	delete[] cpu_constraints;
}
