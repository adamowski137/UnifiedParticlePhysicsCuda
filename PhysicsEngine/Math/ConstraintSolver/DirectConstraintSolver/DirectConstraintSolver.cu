#include "hip/hip_runtime.h"
#include "DirectConstraintSolver.cuh"
#include <hip/hip_runtime.h>
#include "../../../GpuErrorHandling.hpp"
#include <>
#include "../../../Constants.hpp"
#include "../../../Constraint/ConstraintStorage.cuh"

template <typename T>
__global__ void solveConstraintsDirectlyKern(int nConstraints,
	float* x, float* y, float* z,
	float* dx, float* dy, float* dz,
	float* invmass, int* nConstraintsPerParticle,
	T* constraints)
{
	const int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= nConstraints) return;
	constraints[index].directSolve(x, y, z, dx, dy, dz, invmass, nConstraintsPerParticle);
}

__global__ void applyOffset(int nParticles,
	float* x, float* y, float* z,
	float* dx, float* dy, float* dz,
	int* nConstraintsPerParticle)
{
	const int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= nParticles) return;
	const float omega = 1.5f;
	if (nConstraintsPerParticle[index] > 0)
	{
		x[index] += omega * dx[index] / nConstraintsPerParticle[index];
		y[index] += omega * dy[index] / nConstraintsPerParticle[index];
		z[index] += omega * dz[index] / nConstraintsPerParticle[index];
	}
}


DirectConstraintSolver::DirectConstraintSolver(int nParticles) : ConstraintSolver(nParticles)
{
	gpuErrchk(hipMalloc((void**)&dev_nConstraintsPerParticle, nParticles * sizeof(float)));
}

DirectConstraintSolver::~DirectConstraintSolver()
{
	gpuErrchk(hipFree(dev_nConstraintsPerParticle));
}

void DirectConstraintSolver::calculateForces(float* new_x, float* new_y, float* new_z, float* invmass, int* phase, float dt, int iterations)
{
	this->projectConstraints<DistanceConstraint>(new_x, new_y, new_z, invmass, phase, dt, iterations);
	this->projectConstraints<SurfaceConstraint>(new_x, new_y, new_z, invmass, phase, dt, iterations);
	clearAllConstraints();
}

void DirectConstraintSolver::calculateStabilisationForces(float* x, float* y, float* z, int* mode, float* new_x, float* new_y, float* new_z, float* invmass, float dt, int iterations)
{
	throw -1;
}

template<typename T>
void DirectConstraintSolver::projectConstraints(float* x, float* y, float* z, float* invmass, int* phase, float dt, int iterations)
{
	hipMemset(dev_dx, 0, sizeof(float) * nParticles);
	hipMemset(dev_dy, 0, sizeof(float) * nParticles);
	hipMemset(dev_dz, 0, sizeof(float) * nParticles);
	hipMemset(dev_nConstraintsPerParticle, 0, sizeof(int) * nParticles);

	auto constraintData = ConstraintStorage<T>::Instance.getConstraints(true);

	int threads = 32;
	int blocks = (constraintData.second + threads - 1) / threads;
	int particleBlocks = (nParticles + threads - 1) / threads;
	if (constraintData.second > 0)
	{
		solveConstraintsDirectlyKern << <blocks, threads >> > (constraintData.second, x, y, z, dev_dx, dev_dy, dev_dz, invmass, dev_nConstraintsPerParticle, constraintData.first);
		applyOffset << <particleBlocks, threads >> > (nParticles, x, y, z, dev_dx, dev_dy, dev_dz, dev_nConstraintsPerParticle);
	}
}


