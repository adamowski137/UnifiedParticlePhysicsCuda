#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include "LinearSolver.cuh"
#include <cmath>
#include <chrono>
#include <iostream>
#include "../GpuErrorHandling.hpp"

__global__ void jaccobiKern(int n, float* A, float* b, float* x, float* outX, float* c_min, float* c_max)
{
	const int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= n) return;
	float a = A[index * n + index];
	float cx = b[index];
	for (int i = 0; i < n; i++)
	{
		if (i == index) continue;
		cx -= (A[index * n + i] * x[index]);
	}
	//float out = cx / a;
	//out = out > c_min[index] ? out : c_min[index];
	//out = out < c_max[index] ? out : c_max[index];
	outX[index] = min(max(cx / a, c_min[index]), c_max[index]);
	//outX[index] = out;
//	outX[index] = cx / a;
	

}

void jaccobi(int n, float* A, float* b, float* x, float* new_x, float* c_min, float* c_max, int iterations)
{
	int threadsPerBlock = 512;
	int blocks = ceilf((float)n / threadsPerBlock);
	for (int i = 0; i < iterations; i++)
	{
		jaccobiKern << <blocks, threadsPerBlock >> > (n, A, b, x, new_x, c_min, c_max);
		gpuErrchk(hipGetLastError());
		gpuErrchk(hipDeviceSynchronize());

		std::swap(new_x, x);
	}
}