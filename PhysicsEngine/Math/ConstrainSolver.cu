#include "hip/hip_runtime.h"
#include "ConstrainSolver.cuh"
#include "../GpuErrorHandling.hpp"
#include <hip/hip_runtime.h>
#include <>
#include <thrust/transform.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include "../Constants.hpp"

#define SHMEM_SIZE 1024


__global__ void matrixMulKern(const float* a, const float* b, float* c, int N, int K) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ float s_a[SHMEM_SIZE];
	__shared__ float s_b[SHMEM_SIZE];

	float tmp = 0;

	for (int i = 0; i < K; i += blockDim.x)
	{

		s_a[threadIdx.y * blockDim.x + threadIdx.x] = 0;
		s_b[threadIdx.y * blockDim.x + threadIdx.x] = 0;
		__syncthreads();


		if (row < N && i + threadIdx.x < K)
			s_a[threadIdx.y * blockDim.x + threadIdx.x] = a[row * K + i + threadIdx.x];
		if (col < N && i + threadIdx.y < K)
			s_b[threadIdx.y * blockDim.x + threadIdx.x] = b[i * N + threadIdx.y * N + col];
		__syncthreads();

		if (row < N && col < N)
		{
			for (int j = 0; j < blockDim.x; j++) {
				tmp += s_a[threadIdx.y * blockDim.x + j] * s_b[j * blockDim.x + threadIdx.x];
			}
		}
		__syncthreads();
	}

	if (row < N && col < N)
		c[row * N + col] = tmp;
}

__global__ void massVectorMultpilyKern(int columns, int rows, float* invMass, float* J)
{
	const int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= columns * rows) return;
	int column = index % columns;
	J[index] *= invMass[column / 3];
}

__global__ void transposeKern(int columns, int rows, float* A, float* AT)
{
	const int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= columns * rows) return;
	int column = index % columns;
	int row = index / columns;

	AT[column * rows + row] = A[row * columns + column];
}


__global__ void applyForce(float* new_lambda, float* jacobi_transposed, float* fc, int nParticles, int nConstraints)
{
	const int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index < 3 * nParticles)
	{
		for (int i = 0; i < nConstraints; i++)
		{
			fc[index] += new_lambda[i] * jacobi_transposed[index * nConstraints + i];
		}
	}
}

ConstrainSolver::ConstrainSolver(int particles) : nParticles{ particles }
{
	// set pointers to 0 - this way it will be easy to distinguish whether they have already been allocated or not
	dev_jacobian = 0;
	dev_jacobian_transposed = 0;
	dev_velocity_jacobian = 0;
	dev_A = 0;
	dev_b = 0;
	dev_lambda = 0;
	dev_new_lambda = 0;
	dev_c_min = 0;
	dev_c_max = 0;

	ConstrainStorage::Instance.initInstance();
}

ConstrainSolver::~ConstrainSolver()
{
	gpuErrchk(hipFree(dev_jacobian));
	gpuErrchk(hipFree(dev_jacobian_transposed));
	gpuErrchk(hipFree(dev_velocity_jacobian));
	gpuErrchk(hipFree(dev_A));
	gpuErrchk(hipFree(dev_b));
	gpuErrchk(hipFree(dev_lambda));
	gpuErrchk(hipFree(dev_new_lambda));
	gpuErrchk(hipFree(dev_c_min));
	gpuErrchk(hipFree(dev_c_max));
}

void ConstrainSolver::calculateForces(
	float* x, float* y, float* z,
	float* new_x, float* new_y, float* new_z,
	float* vx, float* vy, float* vz,
	float* invmass, float* fc, float dt
)
{
	this->projectConstraints<DistanceConstrain>(fc, invmass, x, y, z, vx, vy, vz, dt, ConstrainType::DISTANCE);
	this->projectConstraints<SurfaceConstraint>(fc, invmass, x, y, z, vx, vy, vz, dt, ConstrainType::SURFACE);
}

void ConstrainSolver::setStaticConstraints(std::vector<std::pair<int, int>> pairs, float d)
{
	std::vector<DistanceConstrain> cpu_constraints;
	for (const auto& pair : pairs)
	{
		cpu_constraints.push_back(DistanceConstrain().init(d, pair.first, pair.second, ConstraintLimitType::EQ));
	}

	ConstrainStorage::Instance.setStaticConstraints<DistanceConstrain>(cpu_constraints.data(), cpu_constraints.size(), ConstrainType::DISTANCE);

}

void ConstrainSolver::addDynamicConstraints(List* collisions, int* sums, float d, ConstraintLimitType type)
{
	ConstrainStorage::Instance.addCollisions(collisions, sums, type, d, nParticles);
}

void ConstrainSolver::addSurfaceConstraints(SurfaceConstraint* surfaceConstraints, int nSurfaceConstraints)
{
	this->dev_surfaceConstraints = surfaceConstraints;
}

void ConstrainSolver::allocateArrays(int nConstraints)
{
	if (nConstraints > nConstraintsMaxAllocated)
	{
		if (dev_jacobian_transposed != 0)
			gpuErrchk(hipFree(dev_jacobian_transposed));
		gpuErrchk(hipMalloc((void**)&dev_jacobian_transposed, 3 * nParticles * nConstraints * sizeof(float)));
		gpuErrchk(hipMemset(dev_jacobian_transposed, 0, 3 * nParticles * nConstraints * sizeof(float)));

		if (dev_velocity_jacobian != 0)
			gpuErrchk(hipFree(dev_velocity_jacobian));
		gpuErrchk(hipMalloc((void**)&dev_velocity_jacobian, 3 * nParticles * nConstraints * sizeof(float)));
		gpuErrchk(hipMemset(dev_velocity_jacobian, 0, 3 * nParticles * nConstraints * sizeof(float)));

		if (dev_A != 0)
			gpuErrchk(hipFree(dev_A));
		gpuErrchk(hipMalloc((void**)&dev_A, nConstraints * nConstraints * sizeof(float)));
		gpuErrchk(hipMemset(dev_A, 0, nConstraints * nConstraints * sizeof(float)));

		if (dev_b != 0)
			gpuErrchk(hipFree(dev_b));
		gpuErrchk(hipMalloc((void**)&dev_b, nConstraints * sizeof(float)));
		gpuErrchk(hipMemset(dev_b, 0, nConstraints * sizeof(float)));

		if (dev_lambda != 0)
			gpuErrchk(hipFree(dev_lambda));
		gpuErrchk(hipMalloc((void**)&dev_lambda, nConstraints * sizeof(float)));
		gpuErrchk(hipMemset(dev_lambda, 0, nConstraints * sizeof(float)));

		if (dev_new_lambda != 0)
			gpuErrchk(hipFree(dev_new_lambda));
		gpuErrchk(hipMalloc((void**)&dev_new_lambda, nConstraints * sizeof(float)));
		gpuErrchk(hipMemset(dev_new_lambda, 0, nConstraints * sizeof(float)));


		if (dev_c_min != 0)
			gpuErrchk(hipFree(dev_c_min));
		gpuErrchk(hipMalloc((void**)&dev_c_min, nConstraints * sizeof(float)));
		gpuErrchk(hipMemset(dev_c_min, 0, nConstraints * sizeof(float)));

		if (dev_c_max != 0)
			gpuErrchk(hipFree(dev_c_max));
		gpuErrchk(hipMalloc((void**)&dev_c_max, nConstraints * sizeof(float)));
		gpuErrchk(hipMemset(dev_c_max, 0, nConstraints * sizeof(float)));

	}
}


