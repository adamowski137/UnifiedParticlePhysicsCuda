#include "hip/hip_runtime.h"
#include "ConstrainSolver.cuh"
#include "../GpuErrorHandling.hpp"
#include <hip/hip_runtime.h>
#include <>
#include <thrust/for_each.h>
#include <thrust/device_ptr.h>
#include "LinearSolver.cuh"

#define SHMEM_SIZE 1024

__global__ void fillJacobiansKern(
	int constrainsAmount, int particles,
	float* x, float* y, float* z,
	float* vx, float* vy, float* vz,
	float* jacobian, float* velocity_jacobian,
	DistanceConstrain* constrains)
{
	const int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= constrainsAmount) return;
	for (int i = 0; i < constrains[index].n; i++)
	{
		constrains[index].positionDerivative(x, y, z, vx, vy, vz, i, &jacobian[index * 3 * particles + 3 * constrains[index].dev_indexes[i]]);
		constrains[index].timePositionDerivative(x, y, z, vx, vy, vz, i, &velocity_jacobian[index * 3 * particles + 3 * constrains[index].dev_indexes[i]]);
	}
}
__global__ void matrixMulKern(const float* a, const float* b, float* c, int N, int K) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ float s_a[SHMEM_SIZE];
	__shared__ float s_b[SHMEM_SIZE];

	float tmp = 0;

	for (int i = 0; i < K; i += blockDim.x)
	{

		s_a[threadIdx.y * blockDim.x + threadIdx.x] = 0;
		s_b[threadIdx.y * blockDim.x + threadIdx.x] = 0;
		__syncthreads();


		if (row < N && col < N)
		{
			if (i + threadIdx.x < K)
				s_a[threadIdx.y * blockDim.x + threadIdx.x] = a[row * K + i + threadIdx.x];
			if (i + threadIdx.y < K)
				s_b[threadIdx.y * blockDim.x + threadIdx.x] = b[i * N + threadIdx.y * N + col];
		}
		__syncthreads();

		if (row < N && col < N)
		{
			for (int j = 0; j < blockDim.x; j++) {
				tmp += s_a[threadIdx.y * blockDim.x + j] * s_b[j * blockDim.x + threadIdx.x];
			}
		}
		__syncthreads();
	}

	if (row < N && col < N)
		c[row * N + col] = tmp;
}

__global__ void massVectorMultpilyKern(int columns, int rows, float* invMass, float* J)
{
	const int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= columns * rows) return;
	int column = index % columns;
	J[index] *= invMass[column / 3];
}

__global__ void transposeKern(int columns, int rows, float* A, float* AT)
{
	const int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= columns * rows) return;
	int column = index % columns;
	int row = index / columns;

	AT[column * rows + row] = A[row * columns + column];
}

ConstrainSolver::ConstrainSolver(int particles, int constrainsNumber) : particles{ particles }, constrainsNumber{constrainsNumber}
{
	int* indexes = new int[2];
	indexes[0] = 0;
	indexes[1] = 1;
	DistanceConstrain* constrains = new DistanceConstrain{ 0.1f, indexes };
	gpuErrchk(hipMalloc((void**)&dev_constrains, constrainsNumber * sizeof(DistanceConstrain)));
	gpuErrchk(hipMemcpy(dev_constrains, constrains, constrainsNumber * sizeof(DistanceConstrain), hipMemcpyHostToDevice));
	delete[] indexes;

	gpuErrchk(hipMalloc((void**)&dev_jacobian, 3 * particles * constrainsNumber * sizeof(float)));
	gpuErrchk(hipMemset(dev_jacobian, 0, 3 * particles * constrainsNumber * sizeof(float)));

	gpuErrchk(hipMalloc((void**)&dev_jacobian_transposed, 3 * particles * constrainsNumber * sizeof(float)));
	gpuErrchk(hipMemset(dev_jacobian_transposed, 0, 3 * particles * constrainsNumber * sizeof(float)));

	gpuErrchk(hipMalloc((void**)&dev_velocity_jacobian, 3 * particles * constrainsNumber * sizeof(float)));
	gpuErrchk(hipMemset(dev_velocity_jacobian, 0, 3 * particles * constrainsNumber * sizeof(float)));

	gpuErrchk(hipMalloc((void**)&dev_A, 3 * particles * 3 * particles * sizeof(float)));
	gpuErrchk(hipMemset(dev_A, 0, 3 * particles * 3 * particles * sizeof(float)));

	gpuErrchk(hipMalloc((void**)&dev_b, 3 * particles * sizeof(float)));
	gpuErrchk(hipMemset(dev_b, 0, 3 * particles * sizeof(float)));

	gpuErrchk(hipMalloc((void**)&dev_lambda, 3 * particles * sizeof(float)));
	gpuErrchk(hipMemset(dev_lambda, 0, 3 * particles * sizeof(float)));

	gpuErrchk(hipMalloc((void**)&dev_new_lambda, 3 * particles * sizeof(float)));
	gpuErrchk(hipMemset(dev_new_lambda, 0, 3 * particles * sizeof(float)));

}

ConstrainSolver::~ConstrainSolver()
{
	gpuErrchk(hipFree(dev_constrains));
	gpuErrchk(hipFree(dev_jacobian));
	gpuErrchk(hipFree(dev_jacobian_transposed));
	gpuErrchk(hipFree(dev_velocity_jacobian));
	gpuErrchk(hipFree(dev_A));
	gpuErrchk(hipFree(dev_b));
	gpuErrchk(hipFree(dev_lambda));
	gpuErrchk(hipFree(dev_new_lambda));
}

void ConstrainSolver::calculateForces(
	float* x, float* y, float* z,
	float* vx, float* vy, float* vz,
	float* invmass
)
{
	unsigned int threads = 32;
	int blocks = ceilf(constrainsNumber / (float)threads);

	fillJacobiansKern << < blocks, threads >> > (constrainsNumber, particles,
		x, y, z,
		vx, vy, vz,
		dev_jacobian, dev_velocity_jacobian,
		dev_constrains);

	gpuErrchk(hipGetLastError());
	gpuErrchk(hipDeviceSynchronize());

	transposeKern << <blocks, threads>> > (
		3 * particles,
		constrainsNumber,
		dev_jacobian,
		dev_jacobian_transposed);

	gpuErrchk(hipGetLastError());
	gpuErrchk(hipDeviceSynchronize());


	massVectorMultpilyKern << <blocks, threads >> > (
		3 * particles,
		constrainsNumber,
		invmass,
		dev_jacobian);

	gpuErrchk(hipGetLastError());
	gpuErrchk(hipDeviceSynchronize());


	unsigned int BLOCKS_X = (3 * particles + threads - 1) / threads;
	unsigned int BLOCKS_Y = (3 * particles + threads - 1) / threads;

	dim3 t{ threads, threads };
	dim3 b{ BLOCKS_X, BLOCKS_Y };

	matrixMulKern<<<b, t>>>(dev_jacobian, dev_jacobian_transposed, dev_A, 3 * particles, constrainsNumber);

	gpuErrchk(hipGetLastError());
	gpuErrchk(hipDeviceSynchronize());


	jaccobiKern << <blocks, threads >> > (3 * particles, dev_A, dev_b, dev_lambda, dev_new_lambda);

	gpuErrchk(hipGetLastError());
	gpuErrchk(hipDeviceSynchronize());

	//std::swap(dev_lambda, dev_new_lambda);

	int N = particles * 3 * particles * 3;
	float* tmp = new float[N];
	hipMemcpy(tmp, dev_A, N * sizeof(float), hipMemcpyDeviceToHost);

	for (int i = 0; i < 3 * particles; i++)
	{
		for (int j = 0; j < 3 * particles; j++)
		{
			std::cout << tmp[i * 3 * particles + j] << " ";
		}
		std::cout << std::endl;
	}
	/*float* tmp = new float[3 * particles * constrainsNumber];
	gpuErrchk(hipMemcpy(tmp, dev_jacobian, 3 * particles * constrainsNumber * sizeof(float), hipMemcpyDeviceToHost));
	for (int j = 0; j < constrainsNumber; j++)
	{
		for (int i = 0; i < particles; i++)
		{
			std::cout << " x: " << tmp[3 * i + j * constrainsNumber] << " y: " << tmp[3 * i + 1 + j * constrainsNumber] << " z: " << tmp[3 * i + 2 + j * constrainsNumber];
		}
		std::cout << "\n";
	}*/

}
