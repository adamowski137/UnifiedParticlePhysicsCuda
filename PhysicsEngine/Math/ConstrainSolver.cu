#include "hip/hip_runtime.h"
#include "ConstrainSolver.cuh"
#include "../GpuErrorHandling.hpp"
#include <hip/hip_runtime.h>
#include <>
#include <thrust/transform.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include "LinearSolver.cuh"
#include "../Constants.hpp"
#include "../Constrain/ConstrainStorage.cuh"

#define SHMEM_SIZE 1024


template<typename T>
__global__ void fillJacobiansKern(
	int nConstraints, int nParticles,
	float* x, float* y, float* z,
	float* vx, float* vy, float* vz,
	float* jacobian, float* velocity_jacobian,
	T* constrains, ConstrainType type)
{
	const int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= nConstraints) return;
	if (type == ConstrainType::DISTANCE)
	{
		(constrains[index]).positionDerivative(x, y, z, vx, vy, vz, 0, &jacobian[index * 3 * nParticles + 3 * (constrains[index]).p[0]]);
		(constrains[index]).timePositionDerivative(x, y, z, vx, vy, vz, 0, &velocity_jacobian[index * 3 * nParticles + 3 * (constrains[index]).p[0]]);

		(constrains[index]).positionDerivative(x, y, z, vx, vy, vz, 1, &jacobian[index * 3 * nParticles + 3 * (constrains[index]).p[1]]);
		(constrains[index]).timePositionDerivative(x, y, z, vx, vy, vz, 1, &velocity_jacobian[index * 3 * nParticles + 3 * (constrains[index]).p[1]]);
	}
	if (type == ConstrainType::SURFACE)
	{
		(constrains[index]).positionDerivative(x, y, z, vx, vy, vz, 0, &jacobian[index * 3 * nParticles + 3 * (constrains[index]).p[0]]);
		(constrains[index]).timePositionDerivative(x, y, z, vx, vy, vz, 0, &velocity_jacobian[index * 3 * nParticles + 3 * (constrains[index]).p[0]]);
	}
}


__global__ void matrixMulKern(const float* a, const float* b, float* c, int N, int K) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ float s_a[SHMEM_SIZE];
	__shared__ float s_b[SHMEM_SIZE];

	float tmp = 0;

	for (int i = 0; i < K; i += blockDim.x)
	{

		s_a[threadIdx.y * blockDim.x + threadIdx.x] = 0;
		s_b[threadIdx.y * blockDim.x + threadIdx.x] = 0;
		__syncthreads();


		if (row < N && i + threadIdx.x < K)
			s_a[threadIdx.y * blockDim.x + threadIdx.x] = a[row * K + i + threadIdx.x];
		if (col < N && i + threadIdx.y < K)
			s_b[threadIdx.y * blockDim.x + threadIdx.x] = b[i * N + threadIdx.y * N + col];
		__syncthreads();

		if (row < N && col < N)
		{
			for (int j = 0; j < blockDim.x; j++) {
				tmp += s_a[threadIdx.y * blockDim.x + j] * s_b[j * blockDim.x + threadIdx.x];
			}
		}
		__syncthreads();
	}

	if (row < N && col < N)
		c[row * N + col] = tmp;
}

__global__ void massVectorMultpilyKern(int columns, int rows, float* invMass, float* J)
{
	const int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= columns * rows) return;
	int column = index % columns;
	J[index] *= invMass[column / 3];
}

__global__ void transposeKern(int columns, int rows, float* A, float* AT)
{
	const int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= columns * rows) return;
	int column = index % columns;
	int row = index / columns;

	AT[column * rows + row] = A[row * columns + column];
}

template <typename T>
__global__ void fillResultVectorKern(int particles, int constrainsNumber, float* b,
	float* x, float* y, float* z,
	float* vx, float* vy, float* vz,
	float* jacobian, float dt,
	float* dev_c_min, float* dev_c_max,
	T* constrains)
{
	const int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= constrainsNumber) return;
	b[index] = -(constrains[index])(x, y, z, vx, vy, vz);
	dev_c_max[index] = constrains[index].cMax;
	dev_c_min[index] = constrains[index].cMin;
}


__global__ void applyForce(float* new_lambda, float* jacobi_transposed, float* fc, int nParticles, int nConstraints)
{
	const int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index < 3 * nParticles)
	{
		for (int i = 0; i < nConstraints; i++)
		{
			fc[index] += new_lambda[i] * jacobi_transposed[index * nConstraints + i];
		}
	}
}

ConstrainSolver::ConstrainSolver(int particles) : nParticles{ particles }
{
	// set pointers to 0 - this way it will be easy to distinguish whether they have already been allocated or not
	dev_jacobian = 0;
	dev_jacobian_transposed = 0;
	dev_velocity_jacobian = 0;
	dev_A = 0;
	dev_b = 0;
	dev_lambda = 0;
	dev_new_lambda = 0;
	dev_c_min = 0;
	dev_c_max = 0;

	nDynamicConstraints = 0;
	nStaticConstraints = 0;
	nSurfaceConstraints = 0;
	nConstraints = 0;

	ConstrainStorage::Instance.initInstance();
}

ConstrainSolver::~ConstrainSolver()
{
	gpuErrchk(hipFree(dev_jacobian));
	gpuErrchk(hipFree(dev_jacobian_transposed));
	gpuErrchk(hipFree(dev_velocity_jacobian));
	gpuErrchk(hipFree(dev_A));
	gpuErrchk(hipFree(dev_b));
	gpuErrchk(hipFree(dev_lambda));
	gpuErrchk(hipFree(dev_new_lambda));
	gpuErrchk(hipFree(dev_c_min));
	gpuErrchk(hipFree(dev_c_max));
}

void ConstrainSolver::calculateForces(
	float* x, float* y, float* z,
	float* new_x, float* new_y, float* new_z,
	float* vx, float* vy, float* vz,
	float* invmass, float* fc, float dt
)
{
	nConstraints = nStaticConstraints + nDynamicConstraints + nSurfaceConstraints;
	if (nConstraints == 0) return;
	//std::cout << "i work" << "\n";

	unsigned int threads = 32;

	// kernels bound by number of constraints
	int constraint_bound_blocks = (nConstraints + threads - 1) / threads;

	// kernels bound by the size of Jacobian
	int jacobian_bound_blocks = ((3 * nParticles * nConstraints) + threads - 1) / threads;

	int particlex3_bound_blocks = ((3 * nParticles) + threads - 1) / threads;

	int particle_bound_blocks = (nParticles + threads - 1) / threads;

	this->allocateArrays();
	this->projectConstraints(x, y, z, vx, vy, vz, dt);


	transposeKern << <jacobian_bound_blocks, threads >> > (
		3 * nParticles,
		nConstraints,
		dev_jacobian,
		dev_jacobian_transposed);

	gpuErrchk(hipGetLastError());
	gpuErrchk(hipDeviceSynchronize())

	massVectorMultpilyKern << <jacobian_bound_blocks, threads >> > (
		3 * nParticles,
		nConstraints,
		invmass,
		dev_jacobian);

	gpuErrchk(hipGetLastError());
	gpuErrchk(hipDeviceSynchronize());

	unsigned int BLOCKS_X = (nConstraints + threads - 1) / threads;
	unsigned int BLOCKS_Y = (nConstraints + threads - 1) / threads;

	dim3 t{ threads, threads };
	dim3 b{ BLOCKS_X, BLOCKS_Y };

	matrixMulKern << <b, t >> > (dev_jacobian, dev_jacobian_transposed, dev_A, nConstraints, 3 * nParticles);

	gpuErrchk(hipGetLastError());
	gpuErrchk(hipDeviceSynchronize());

	jaccobi(nConstraints, dev_A, dev_b, dev_lambda, dev_new_lambda, dev_c_min, dev_c_max, 1);

	applyForce << <particlex3_bound_blocks, threads >> > (dev_new_lambda, dev_jacobian_transposed, fc, nParticles, nConstraints);

	gpuErrchk(hipGetLastError());
	gpuErrchk(hipDeviceSynchronize());
}

void ConstrainSolver::setStaticConstraints(std::vector<std::pair<int, int>> pairs, float d)
{
	nStaticConstraints = pairs.size();
	std::vector<DistanceConstrain> cpu_constraints;
	for (const auto& pair : pairs)
	{
		cpu_constraints.push_back(DistanceConstrain().init(d, pair.first, pair.second, ConstraintLimitType::EQ));
	}

	ConstrainStorage::Instance.setStaticConstraints<DistanceConstrain>(cpu_constraints.data(), cpu_constraints.size(), ConstrainType::DISTANCE);

}

void ConstrainSolver::addDynamicConstraints(List* collisions, int* sums, float d, ConstraintLimitType type)
{
	ConstrainStorage::Instance.addCollisions(collisions, sums, type, d, nParticles);
}

void ConstrainSolver::addSurfaceConstraints(SurfaceConstraint* surfaceConstraints, int nSurfaceConstraints)
{
	this->dev_surfaceConstraints = surfaceConstraints;
	this->nSurfaceConstraints = nSurfaceConstraints;
}

void ConstrainSolver::allocateArrays()
{
	if (nConstraints > nConstraintsMaxAllocated)
	{
		if (dev_jacobian_transposed != 0)
			gpuErrchk(hipFree(dev_jacobian_transposed));
		gpuErrchk(hipMalloc((void**)&dev_jacobian_transposed, 3 * nParticles * nConstraints * sizeof(float)));
		gpuErrchk(hipMemset(dev_jacobian_transposed, 0, 3 * nParticles * nConstraints * sizeof(float)));

		if (dev_velocity_jacobian != 0)
			gpuErrchk(hipFree(dev_velocity_jacobian));
		gpuErrchk(hipMalloc((void**)&dev_velocity_jacobian, 3 * nParticles * nConstraints * sizeof(float)));
		gpuErrchk(hipMemset(dev_velocity_jacobian, 0, 3 * nParticles * nConstraints * sizeof(float)));

		if (dev_A != 0)
			gpuErrchk(hipFree(dev_A));
		gpuErrchk(hipMalloc((void**)&dev_A, nConstraints * nConstraints * sizeof(float)));
		gpuErrchk(hipMemset(dev_A, 0, nConstraints * nConstraints * sizeof(float)));

		if (dev_b != 0)
			gpuErrchk(hipFree(dev_b));
		gpuErrchk(hipMalloc((void**)&dev_b, nConstraints * sizeof(float)));
		gpuErrchk(hipMemset(dev_b, 0, nConstraints * sizeof(float)));

		if (dev_lambda != 0)
			gpuErrchk(hipFree(dev_lambda));
		gpuErrchk(hipMalloc((void**)&dev_lambda, nConstraints * sizeof(float)));
		gpuErrchk(hipMemset(dev_lambda, 0, nConstraints * sizeof(float)));

		if (dev_new_lambda != 0)
			gpuErrchk(hipFree(dev_new_lambda));
		gpuErrchk(hipMalloc((void**)&dev_new_lambda, nConstraints * sizeof(float)));
		gpuErrchk(hipMemset(dev_new_lambda, 0, nConstraints * sizeof(float)));


		if (dev_c_min != 0)
			gpuErrchk(hipFree(dev_c_min));
		gpuErrchk(hipMalloc((void**)&dev_c_min, nConstraints * sizeof(float)));
		gpuErrchk(hipMemset(dev_c_min, 0, nConstraints * sizeof(float)));

		if (dev_c_max != 0)
			gpuErrchk(hipFree(dev_c_max));
		gpuErrchk(hipMalloc((void**)&dev_c_max, nConstraints * sizeof(float)));
		gpuErrchk(hipMemset(dev_c_max, 0, nConstraints * sizeof(float)));

	}
}


void ConstrainSolver::projectConstraints(float* x, float* y, float* z, float* vx, float* vy, float* vz, float dt)
{
	int threads = 32;
	int blocks = (nConstraints + threads - 1) / threads;
	std::pair<DistanceConstrain*, int> distanceConstrains = ConstrainStorage::Instance.getConstraints<DistanceConstrain>(ConstrainType::DISTANCE);
	fillJacobiansKern<DistanceConstrain> << < blocks, threads >> > (distanceConstrains.second, nParticles,
		x, y, z,
		vx, vy, vz,
		dev_jacobian, dev_velocity_jacobian,
		distanceConstrains.first, ConstrainType::DISTANCE);

	gpuErrchk(hipGetLastError());
	gpuErrchk(hipDeviceSynchronize());

	fillResultVectorKern<DistanceConstrain> << <blocks, threads >> > (nParticles, distanceConstrains.second, dev_b,
		x, y, z,
		vx, vy, vz, dev_jacobian, dt,
		dev_c_min, dev_c_max,
		distanceConstrains.first);

	gpuErrchk(hipGetLastError());
	gpuErrchk(hipDeviceSynchronize());

	if (nSurfaceConstraints > 0)
	{
		std::pair<SurfaceConstraint*, int> surfaceConstrains = ConstrainStorage::Instance.getConstraints<SurfaceConstraint>(ConstrainType::SURFACE);
		int offset = distanceConstrains.second;
		fillJacobiansKern<SurfaceConstraint><< < blocks, threads >> > (surfaceConstrains.second, nParticles,
			x + 3 * offset, y + 3 * offset, z + 3 * offset,
			vx + 3 * offset, vy + 3 * offset, vz + 3 * offset,
			dev_jacobian + 3 * nParticles * offset, dev_velocity_jacobian + 3 * nParticles * offset,
			surfaceConstrains.first, ConstrainType::SURFACE);

		gpuErrchk(hipGetLastError());
		gpuErrchk(hipDeviceSynchronize());


		fillResultVectorKern<SurfaceConstraint> << <blocks, threads >> > (nParticles, surfaceConstrains.second, dev_b + offset,
			x + 3 * offset, y + 3 * offset, z + 3 * offset,
			vx + 3 * offset, vy + 3 * offset, vz + 3 * offset, dev_jacobian, dt,
			dev_c_min, dev_c_max,
			surfaceConstrains.first);

		gpuErrchk(hipGetLastError());
		gpuErrchk(hipDeviceSynchronize());


		nSurfaceConstraints = 0;
		gpuErrchk(hipFree(dev_surfaceConstraints));

	}

	nDynamicConstraints = 0;
}
