#include "hip/hip_runtime.h"
#include <glad/glad.h>
#include "Particle.cuh"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstdlib>
#include <cuda_gl_interop.h>
#include <>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}
__global__ void copyKernel(int amount, float* src, float* dst)
{
	const int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= amount) return;
	dst[index] = src[index];
}

__global__ void setDiagonalMatrix(int amount, float* src, float* dst)
{
	const int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= amount) return;
	dst[amount * index + index] = src[index];
}

ParticleType::ParticleType(int amount, float mass) : amountOfParticles{amount}
{
	blocks = ceilf((float)amountOfParticles / THREADS);

	setupDeviceData();
}

ParticleType::~ParticleType()
{
	gpuErrchk(hipFree(dev_x));
	gpuErrchk(hipFree(dev_y));
	gpuErrchk(hipFree(dev_z));
	gpuErrchk(hipFree(dev_vx));
	gpuErrchk(hipFree(dev_vy));
	gpuErrchk(hipFree(dev_vz));
}

void ParticleType::setupDeviceData()
{
	gpuErrchk(hipMalloc((void**)&dev_x, amountOfParticles * sizeof(float)));
	gpuErrchk(hipMalloc((void**)&dev_y, amountOfParticles * sizeof(float)));
	gpuErrchk(hipMalloc((void**)&dev_z, amountOfParticles * sizeof(float)));
	gpuErrchk(hipMalloc((void**)&dev_vx, amountOfParticles * sizeof(float)));
	gpuErrchk(hipMalloc((void**)&dev_vy, amountOfParticles * sizeof(float)));
	gpuErrchk(hipMalloc((void**)&dev_vz, amountOfParticles * sizeof(float)));
	gpuErrchk(hipMalloc((void**)&dev_invmass, amountOfParticles * sizeof(float)));
	gpuErrchk(hipMalloc((void**)&dev_invM, amountOfParticles * amountOfParticles * sizeof(float)));

	setDiagonalMatrix << <THREADS, blocks >> > (amountOfParticles, dev_invmass, dev_invM);

	gpuErrchk(hipGetLastError());
	gpuErrchk(hipDeviceSynchronize());
}

void ParticleType::setupShaderData()
{
	glGenVertexArrays(1, &vao);
	glGenBuffers(1, &vboSphere);
	glGenBuffers(1, &vbox);
	glGenBuffers(1, &vboy);
	glGenBuffers(1, &vboz);
	glBindVertexArray(vao);

	glBindBuffer(GL_ARRAY_BUFFER, vbox);
	glBufferData(GL_ARRAY_BUFFER, (amountOfParticles) * sizeof(GLfloat), nullptr, GL_STREAM_DRAW);
	glEnableVertexAttribArray(0);
	glVertexAttribPointer((GLuint)0, 1, GL_FLOAT, GL_FALSE, 0, 0);

	glBindBuffer(GL_ARRAY_BUFFER, vboy);
	glBufferData(GL_ARRAY_BUFFER, (amountOfParticles) * sizeof(GLfloat), nullptr, GL_STREAM_DRAW);
	glEnableVertexAttribArray(1);
	glVertexAttribPointer((GLuint)1, 1, GL_FLOAT, GL_FALSE, 0, 0);

	glBindBuffer(GL_ARRAY_BUFFER, vboy);
	glBufferData(GL_ARRAY_BUFFER, (amountOfParticles) * sizeof(GLfloat), nullptr, GL_STREAM_DRAW);
	glEnableVertexAttribArray(2);
	glVertexAttribPointer((GLuint)1, 1, GL_FLOAT, GL_FALSE, 0, 0);

	gpuErrchk(cudaGLRegisterBufferObject(vboSphere));
	gpuErrchk(cudaGLRegisterBufferObject(vbox));
	gpuErrchk(cudaGLRegisterBufferObject(vboy));
	gpuErrchk(cudaGLRegisterBufferObject(vboz));
}

void ParticleType::renderData()
{
	float* x, * y, * z;
	cudaGLMapBufferObject((void**)&x, vbox);
	cudaGLMapBufferObject((void**)&y, vboy);
	cudaGLMapBufferObject((void**)&z, vboz);

	copyKernel<<<blocks, THREADS>>>(amountOfParticles, dev_x, x);
	copyKernel<<<blocks, THREADS>>>(amountOfParticles, dev_y, y);
	copyKernel<<<blocks, THREADS>>>(amountOfParticles, dev_z, z);

	gpuErrchk(hipGetLastError());
	gpuErrchk(hipDeviceSynchronize());
	
	cudaGLUnmapBufferObject(vbox);
	cudaGLUnmapBufferObject(vboy);
	cudaGLUnmapBufferObject(vboz);

	glBindVertexArray(vao);
}