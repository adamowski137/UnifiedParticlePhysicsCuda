#include "hip/hip_runtime.h"
#include <glad/glad.h>
#include "Particle.cuh"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstdlib>
#include <cuda_gl_interop.h>
#include <>

#define EPS 0.000001

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

__global__ void initializeRandomKern(int amount, hiprandState* state)
{
	const int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= amount) return;
	hiprand_init(1234, index, 0, &state[index]);
}

__global__ void fillRandomKern(int amount, float* dst, hiprandState* state)
{
	const int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= amount) return;
	dst[index] = 60 * hiprand_uniform(&state[index]) - 30.0f;
}

__global__ void copyToVBOKernel(int amount, float* x, float* y, float* z, float* dst)
{
	const int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= amount) return;
	dst[3 * index + 0] = x[index];
	dst[3 * index + 1] = y[index];
	dst[3 * index + 2] = z[index];
}

__global__ void setDiagonalMatrix(int amount, float* src, float* dst)
{
	const int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= amount) return;
	dst[amount * index + index] = src[index];
}

__global__ void predictPositionsKern(int amount, 
	float* x, float* y, float* z,
	float* new_x, float* new_y, float* new_z, 
	float* vx, float* vy, float* vz,
	float dvx, float dvy, float dvz,
	float dt
)
{
	const int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= amount) return;

	// update velocities
	vx[index] += dvx;
	vy[index] += dvy;
	vz[index] += dvz;

	// predict new position - not the actual new positions
	new_x[index] = x[index] + dt * vx[index];
	new_y[index] = y[index] + dt * vy[index];
	new_z[index] = z[index] + dt * vz[index];

	// apply mass scaling??
}

__global__ void applyChangesKern(int amount,
	float* x, float* y, float* z,
	float* new_x, float* new_y, float* new_z,
	float* vx, float* vy, float* vz, float invdt)
{
	const int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= amount) return;

	float changeX = (new_x[index] - x[index]);
	float changeY = (new_y[index] - y[index]);
	float changeZ = (new_z[index] - z[index]);

	// update velocity
	vx[index] = invdt * (changeX);
	vy[index] = invdt * (changeY);
	vz[index] = invdt * (changeZ);

	// advect diffuse particles ??
	
	// apply internal forces

	// update position or apply sleeping

	float changeSQ = changeX * changeX + changeY * changeY + changeZ * changeZ;
	if (changeSQ > EPS)
	{
		x[index] = new_x[index];
		y[index] = new_y[index];
		z[index] = new_z[index];
	}

}

ParticleType::ParticleType(int amount) : amountOfParticles{amount}
{
	blocks = ceilf((float)amountOfParticles / THREADS);

	setupDeviceData();
}

ParticleType::~ParticleType()
{
	gpuErrchk(hipFree(dev_x));
	gpuErrchk(hipFree(dev_y));
	gpuErrchk(hipFree(dev_z));
	gpuErrchk(hipFree(dev_vx));
	gpuErrchk(hipFree(dev_vy));
	gpuErrchk(hipFree(dev_vz));
}

void ParticleType::setupDeviceData()
{
	gpuErrchk(hipMalloc((void**)&dev_curand, amountOfParticles * sizeof(hiprandState)));
	gpuErrchk(hipMalloc((void**)&dev_x, amountOfParticles * sizeof(float)));
	gpuErrchk(hipMalloc((void**)&dev_y, amountOfParticles * sizeof(float)));
	gpuErrchk(hipMalloc((void**)&dev_z, amountOfParticles * sizeof(float)));

	gpuErrchk(hipMalloc((void**)&dev_new_x, amountOfParticles * sizeof(float)));
	gpuErrchk(hipMalloc((void**)&dev_new_y, amountOfParticles * sizeof(float)));
	gpuErrchk(hipMalloc((void**)&dev_new_z, amountOfParticles * sizeof(float)));
	
	gpuErrchk(hipMalloc((void**)&dev_vx, amountOfParticles * sizeof(float)));
	gpuErrchk(hipMalloc((void**)&dev_vy, amountOfParticles * sizeof(float)));
	gpuErrchk(hipMalloc((void**)&dev_vz, amountOfParticles * sizeof(float)));
	
	gpuErrchk(hipMalloc((void**)&dev_invmass, amountOfParticles * sizeof(float)));
	gpuErrchk(hipMalloc((void**)&dev_invM, amountOfParticles * amountOfParticles * sizeof(float)));

	setDiagonalMatrix << <THREADS, blocks >> > (amountOfParticles, dev_invmass, dev_invM);
	gpuErrchk(hipGetLastError());
	gpuErrchk(hipDeviceSynchronize());

	initializeRandomKern << <THREADS, blocks >> > (amountOfParticles, dev_curand);
	gpuErrchk(hipGetLastError());
	gpuErrchk(hipDeviceSynchronize());

	fillRandomKern << <THREADS, blocks >> > (amountOfParticles, dev_x, dev_curand);
	gpuErrchk(hipGetLastError());
	gpuErrchk(hipDeviceSynchronize());

	fillRandomKern << <THREADS, blocks >> > (amountOfParticles, dev_y, dev_curand);
	gpuErrchk(hipGetLastError());
	gpuErrchk(hipDeviceSynchronize());
	
	fillRandomKern << <THREADS, blocks >> > (amountOfParticles, dev_z, dev_curand);
	gpuErrchk(hipGetLastError());
	gpuErrchk(hipDeviceSynchronize());
}

void ParticleType::renderData(unsigned int vbo)
{
	float* dst;
	cudaGLMapBufferObject((void**)&dst, vbo);

	copyToVBOKernel <<<blocks, THREADS>>>(amountOfParticles, dev_x, dev_y, dev_z, dst);

	gpuErrchk(hipGetLastError());
	gpuErrchk(hipDeviceSynchronize());
	
	cudaGLUnmapBufferObject(vbo);
}

void ParticleType::calculateNewPositions(float dt)
{
	// predict new positions and update velocities
	fextx = 0.0f;
	fexty = -10.0f;
	fextz = 0.0f;

	float dvx = fextx * dt;
	float dvy = fexty * dt;
	float dvz = fextz * dt;

		predictPositionsKern << <THREADS, blocks >> > (
		amountOfParticles,
		dev_x, dev_y, dev_z,
		dev_new_x, dev_new_y, dev_new_z,
		dev_vx, dev_vy, dev_vz,
		dvx, dvy, dvz, dt
		);

	gpuErrchk(hipGetLastError());
	gpuErrchk(hipDeviceSynchronize());

	// find neighboring particles and solid contacts ??

	// todo implement grid (predicted positions)

	// stabilization iterations

	// todo solve contact constrains
	// update predicted position and current positions

	// solve iterations

	// todo solve every constraint group 
	// update predicted position
	applyChangesKern << <THREADS, blocks >> > (
		amountOfParticles,
		dev_x, dev_y, dev_z,
		dev_new_x, dev_new_y, dev_new_z,
		dev_vx, dev_vy, dev_vz,
		1/dt
		);
	gpuErrchk(hipGetLastError());
	gpuErrchk(hipDeviceSynchronize());
}

void ParticleType::mapCudaVBO(unsigned int vbo)
{
	cudaGLRegisterBufferObject(vbo);
}