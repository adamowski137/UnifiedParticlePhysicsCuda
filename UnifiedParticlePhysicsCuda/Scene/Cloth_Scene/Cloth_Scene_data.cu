#include "Cloth_Scene_data.cuh"
#include "../../PhysicsEngine/Particle/ParticleData.cuh"
#include "../../PhysicsEngine/GpuErrorHandling.hpp"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "../../PhysicsEngine/Cloth/Cloth.hpp"

void initData_ClothScene(int nParticles,
	float* dev_x, float* dev_y, float* dev_z,
	float* dev_vx, float* dev_vy, float* dev_vz)
{
	hiprandState* dev_curand;
	int threads = 32;

	int blocks = (nParticles + threads - 1) / threads;
	gpuErrchk(hipMalloc((void**)&dev_curand, nParticles * sizeof(hiprandState)));

	initializeRandomKern << < blocks, threads >> > (nParticles, dev_curand);
	gpuErrchk(hipGetLastError());
	gpuErrchk(hipDeviceSynchronize());

	float d = 3;
	int W = 3;
	int H = 4;
	Cloth::initClothSimulation(H, W, d, -d * W / 2.f, 0.f, 0.f, dev_x, dev_y, dev_z);

	
	fillRandomKern << <blocks, threads >> > (nParticles, dev_z, dev_curand, 0.f, 0.f);
	gpuErrchk(hipGetLastError());
	gpuErrchk(hipDeviceSynchronize());

	fillRandomKern << <blocks, threads >> > (nParticles, dev_vx, dev_curand, -0.f, 0.f);
	gpuErrchk(hipGetLastError());
	gpuErrchk(hipDeviceSynchronize());

	fillRandomKern << <blocks, threads >> > (nParticles, dev_vy, dev_curand, 0.f, 0.f);
	gpuErrchk(hipGetLastError());
	gpuErrchk(hipDeviceSynchronize());

}