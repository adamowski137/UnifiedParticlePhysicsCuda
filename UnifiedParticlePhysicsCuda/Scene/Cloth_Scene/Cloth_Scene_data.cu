#include "Cloth_Scene_data.cuh"
#include "../../PhysicsEngine/Particle/ParticleData.cuh"
#include "../../PhysicsEngine/GpuErrorHandling.hpp"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "../../PhysicsEngine/Cloth/Cloth.hpp"
#include <vector>

void initData_ClothScene(int nParticles,
	float* dev_x, float* dev_y, float* dev_z,
	float* dev_vx, float* dev_vy, float* dev_vz, int* mode)
{
	hiprandState* dev_curand;
	int threads = 32;

	int blocks = (nParticles + threads - 1) / threads;
	gpuErrchk(hipMalloc((void**)&dev_curand, nParticles * sizeof(hiprandState)));

	initializeRandomKern << < blocks, threads >> > (nParticles, dev_curand);
	gpuErrchk(hipGetLastError());
	gpuErrchk(hipDeviceSynchronize());

	float d = 3;
	int W = 10;
	int H = 12;
	Cloth::initClothSimulation(H, W, d, -d * W / 2.f, 0.f, 0.f, dev_x, dev_y, dev_z);

	std::vector<int> modes(nParticles, 0);
	//for (int i = 0; i < W; i++)
	//	modes[i] = 1;
	modes[0] = 1;
	modes[W - 1] = 1;
	gpuErrchk(hipMemcpy(mode, modes.data(), modes.size() * sizeof(int), hipMemcpyHostToDevice));




	
	fillRandomKern << <blocks, threads >> > (nParticles, dev_z, dev_curand, 0.f, 0.f);
	gpuErrchk(hipGetLastError());
	gpuErrchk(hipDeviceSynchronize());

	fillRandomKern << <blocks, threads >> > (nParticles, dev_vx, dev_curand, -0.f, 0.f);
	gpuErrchk(hipGetLastError());
	gpuErrchk(hipDeviceSynchronize());

	fillRandomKern << <blocks, threads >> > (nParticles, dev_vy, dev_curand, 0.f, 0.f);
	gpuErrchk(hipGetLastError());
	gpuErrchk(hipDeviceSynchronize());

}