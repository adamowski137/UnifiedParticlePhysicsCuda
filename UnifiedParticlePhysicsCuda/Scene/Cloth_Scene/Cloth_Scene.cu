#include "Cloth_Scene.cuh"
#include "../../ResourceManager/ResourceManager.hpp"
#include "../../PhysicsEngine/Cloth/Cloth.hpp"

#include "../../PhysicsEngine/Particle/ParticleData.cuh"
#include "../../PhysicsEngine/GpuErrorHandling.hpp"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <vector>

#define CLOTH_W 10
#define CLOTH_H 12

Cloth_Scene::Cloth_Scene() :
	Scene(ResourceManager::Instance.Shaders["instancedphong"], CLOTH_W * CLOTH_H, ANY_CONSTRAINTS_ON)
{
	std::vector<float> offsets;
	offsets.resize(CLOTH_W * CLOTH_H * 3, 0.0f);

	renderer->setSphereScale(0.1f);

	sceneSphere.addInstancing(offsets);
	particles.mapCudaVBO(sceneSphere.instancingVBO);
	particles.setExternalForces(0.f, -9.81f, -20.f);

	camera.setPosition(glm::vec3(0, 0, -10));

	applySceneSetup();
}

Cloth_Scene::~Cloth_Scene()
{
}

void Cloth_Scene::update(float dt)
{
	ConstraintStorage<DistanceConstraint>::Instance.setDynamicConstraints(Cloth::getConstraints().first, Cloth::getConstraints().second);
	particles.calculateNewPositions(dt);
	this->handleKeys();

	renderer->getShader().setUniformMat4fv("VP", camera.getProjectionViewMatrix());
	renderer->setCameraPosition(camera.getPosition());
	renderer->setLightSourcePosition(glm::vec3(0, 0, -10));
}

void Cloth_Scene::draw()
{
	particles.renderData(sceneSphere.instancingVBO);
	renderer->drawInstanced(sceneSphere, particles.particleCount());
}

void Cloth_Scene::initData(int nParticles, float* dev_x, float* dev_y, float* dev_z, float* dev_vx, float* dev_vy, float* dev_vz, int* dev_phase, float* dev_invmass)
{
	hiprandState* dev_curand;
	int threads = 32;

	int blocks = (nParticles + threads - 1) / threads;
	gpuErrchk(hipMalloc((void**)&dev_curand, nParticles * sizeof(hiprandState)));

	initializeRandomKern << < blocks, threads >> > (nParticles, dev_curand);
	gpuErrchk(hipGetLastError());
	gpuErrchk(hipDeviceSynchronize());

	float d = 2;
	int W = CLOTH_W;
	int H = CLOTH_H;
	Cloth::initClothSimulation(H, W, d, -d * W / 2.f, 0.f, 0.f, dev_x, dev_y, dev_z);

	std::vector<float> invmass(nParticles, 1.f);
	invmass[0] = 0.f;
	invmass[W - 1] = 0.f;
	gpuErrchk(hipMemcpy(dev_invmass, invmass.data(), invmass.size() * sizeof(float), hipMemcpyHostToDevice));


	fillRandomKern << <blocks, threads >> > (nParticles, dev_z, dev_curand, 0.f, 0.f);
	gpuErrchk(hipGetLastError());
	gpuErrchk(hipDeviceSynchronize());

	fillRandomKern << <blocks, threads >> > (nParticles, dev_vx, dev_curand, 0.f, 0.f);
	gpuErrchk(hipGetLastError());
	gpuErrchk(hipDeviceSynchronize());

	fillRandomKern << <blocks, threads >> > (nParticles, dev_vy, dev_curand, 0.f, 0.f);
	gpuErrchk(hipGetLastError());
	gpuErrchk(hipDeviceSynchronize());

	fillRandomKern << <blocks, threads >> > (nParticles, dev_vz, dev_curand, 0.f, 0.f);
	gpuErrchk(hipGetLastError());
	gpuErrchk(hipDeviceSynchronize());
}
